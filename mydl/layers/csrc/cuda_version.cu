// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace mydl {
int get_cudart_version() {
  return CUDART_VERSION;
}
} // namespace mydl
